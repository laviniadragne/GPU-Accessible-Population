#include "hip/hip_runtime.h"
#include <math.h>
#include <fstream>
#include <bits/stdc++.h>

#include "helper.h"

#define MAX_CITIES		(1024 * 1024)

using namespace std;

// Fiecare thread calculeaza si stocheaza sinusul si cosinusul pentru
// latitudini si longitudini, pentru calculul ulterior de distante
// intre orase
__global__ void calcSinCos(float *lat, float *lon, float* sin_angle, float* cos_angle, 
    float* sin_angle_90, float* cos_angle_90, int N) {

    register unsigned int i = threadIdx.x + blockDim.x * blockIdx.x;

    if (i < N) {
        register float phi = (90.f - lat[i]) * DEGREE_TO_RADIANS;
        register float theta = lon[i] * DEGREE_TO_RADIANS;

        sin_angle[i] = sin(theta);
        cos_angle[i] = sqrt (1 - sin_angle[i] * sin_angle[i]);
        sin_angle_90[i] = sin(phi);
        cos_angle_90[i] = sqrt (1 - sin_angle_90[i] * sin_angle_90[i]);
    }
}

__global__ void compareDist(float *lat, float *lon,
                            unsigned long long int *pop, unsigned long long int *copy_pop,
                            int kmRange, int N, float* sin_angle, float* cos_angle, 
                            float* sin_angle_90, float* cos_angle_90) 
{
    register unsigned int i = threadIdx.x + blockDim.x * blockIdx.x;

    register float res;

    if (i < N) {
        register int j;

        register unsigned long long int *pop_i = &pop[i];
        register unsigned long long int copy_pop_i = copy_pop[i];
        j = i + 1;
        register unsigned long long int *pop_j = &pop[j];
        register unsigned long long int *copy_pop_j = &copy_pop[j];
        register float *lat_j = &lat[j];
        register float *lon_j = &lon[j]; 

        register float sin_angle_90_var = sin_angle_90[i];
        register float cos_angle_90_var = cos_angle_90[i];
        register float sin_angle_var = sin_angle[i];
        register float cos_angle_var = cos_angle[i];
    
        j = i + 1;
        register float *sin_angle_90_ptr = &sin_angle_90[j];
        register float *cos_angle_90_ptr = &cos_angle_90[j];
        register float *sin_angle_ptr = &sin_angle[j];
        register float *cos_angle_ptr = &cos_angle[j];
    
        // Calculez toate distantele de la orasul i la cele mai mari
        // decat el in lista de orase
        for (j = i + 1; j < N; j++) {
            // Calculul distantei efective pe baza latitudinii si longitudinii
            register float cs = sin_angle_90_var * (*sin_angle_90_ptr) * (cos_angle_var * (*cos_angle_ptr) + 
                                sin_angle_var * (*sin_angle_ptr)) + cos_angle_90_var * (*cos_angle_90_ptr);
            if (cs > 1) {
                cs = 1;
            } else if (cs < -1) {
                cs = -1;
            }
        
            res = 6371.f * acos(cs);

            if (res <= kmRange) {
                atomicAdd(pop_i, *copy_pop_j);
                atomicAdd(pop_j, copy_pop_i);
            }
            pop_j++;
            copy_pop_j++;
            lat_j++;
            lon_j++;

            sin_angle_90_ptr++;
            cos_angle_90_ptr++;
            sin_angle_ptr++;
            cos_angle_ptr++;
        }
    }
}

// sampleFileIO demos reading test files and writing output
void sampleFileIO(float kmRange, const char* fileIn, const char* fileOut)
{
    register string geon;
    register float lat, lon;
    register int pop;
    register float *device_lat = 0;
    register float *device_lon = 0;
    register float *cos_angle = 0;
    register float *sin_angle = 0;
    register float *cos_angle_90 = 0;
    register float *sin_angle_90 = 0;
    register unsigned long long int *device_pop = 0;
    register unsigned long long int *copy_pop_device;
    register float *host_lat = 0;
    register float *host_lon = 0;
    register unsigned long long int *host_pop = 0;
    register unsigned long long int N = 0;
    register int i;

    // Aloc datele pentru host
    host_lat = (float *) malloc(MAX_CITIES * sizeof(float));
    host_lon = (float *) malloc(MAX_CITIES * sizeof(float));
    host_pop = (unsigned long long int *) malloc(MAX_CITIES * sizeof(unsigned long long int));

    // Aloc datele pentru device
    hipMalloc((void **) &device_lat, MAX_CITIES * sizeof(float));
    hipMalloc((void **) &device_lon, MAX_CITIES * sizeof(float));
    hipMalloc((void **) &device_pop, MAX_CITIES * sizeof(unsigned long long int));
    hipMalloc((void **) &copy_pop_device, MAX_CITIES * sizeof(unsigned long long int));

    if (host_lat == 0 || host_lon == 0 || host_pop == 0 ||
        device_lat == 0 || device_lon == 0 || device_pop == 0 ||
        copy_pop_device == 0) {
        printf("[*] Error!\n");
        return;
    }

    ifstream ifs(fileIn);
    ofstream ofs(fileOut);

    // Memorez datele de intrare
    while(ifs >> geon >> lat >> lon >> pop)
    {
        host_pop[N] = pop;
        host_lat[N] = lat;
        host_lon[N] = lon;

        N++;
    }

    // Copiez datele din host in device (din CPU in GPU)
    hipMemcpy(device_lat, host_lat, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(device_lon, host_lon, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(device_pop, host_pop, N * sizeof(unsigned long long int), hipMemcpyHostToDevice);
    hipMemcpy(copy_pop_device, host_pop, N * sizeof(unsigned long long int), hipMemcpyHostToDevice);

    // Aloc memorie pentru vectorii in care voi retine sinusul si cosinusul
    // pentru latitudini si longitudini
    hipMalloc((void **) &cos_angle, N * sizeof(unsigned long long int));
    hipMalloc((void **) &sin_angle, N * sizeof(unsigned long long int));
    hipMalloc((void **) &cos_angle_90, N * sizeof(unsigned long long int));
    hipMalloc((void **) &sin_angle_90, N * sizeof(unsigned long long int));

    if (cos_angle == 0 || sin_angle == 0 ||
        cos_angle_90 == 0 || sin_angle_90 == 0) {
        printf("[*] Error!\n");
        return;
    }

    // Calculez numarul de blocuri de care am nevoie
    // pentru a lansa N thread-uri
    register const size_t block_size = 256;
    register size_t num_blocks = N / block_size;

    if (N % block_size) 
    ++num_blocks;

    // Calculez sin si cos necesare
    calcSinCos<<<num_blocks, block_size>>>(device_lat, device_lon, sin_angle, cos_angle,
         sin_angle_90, cos_angle_90, N);
    hipDeviceSynchronize();
    if (hipSuccess != hipGetLastError()) {
        printf("[*] Error!\n");
        return;
    }

    // Calculez distantele si actualizez populatiile in functie de ele
    compareDist<<<num_blocks, block_size>>>(device_lat, device_lon, device_pop,
                copy_pop_device, kmRange, N, sin_angle, cos_angle,
                sin_angle_90, cos_angle_90);

    hipDeviceSynchronize();
    if (hipSuccess != hipGetLastError()) {
        printf("[*] Error!\n");
        return;
    }

    // Copiez populatiile din device in host
    hipMemcpy(host_pop, device_pop, N * sizeof(unsigned long long int), hipMemcpyDeviceToHost);

    // Scriu in fisier populatiile
    for (i = 0; i < N; i++) {
        ofs << host_pop[i] << endl;
    }

    ifs.close();
    ofs.close();

    free(host_lon);
    free(host_lat);
    free(host_pop);

    hipFree(device_lon);
    hipFree(device_lat);
    hipFree(device_pop);
    hipFree(copy_pop_device);
    hipFree(cos_angle);
    hipFree(sin_angle);
    hipFree(cos_angle_90);
    hipFree(sin_angle_90);
}
